// Test program to test if your GPGPU is capable of using curand (and kernel-side printf)
// Compile with: nvcc curand.cu -arch=sm_20
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
 
 
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)
 
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)
 
__global__ void generate_kernel(hiprandStateMtgp32 *state)
{
    for(int i = 0; i < 1000; i++) {
        printf("%f, ", hiprand_normal(&state[blockIdx.x]));
    }
}
 
int main(int argc, char *argv[])
{
    int i;
    hiprandStateMtgp32 *devMTGPStates;
    mtgp32_kernel_params *devKernelParams;
        
    /* Allocate space for prng states on device */
    CUDA_CALL(hipMalloc((void **)&devMTGPStates, 64 * 
              sizeof(hiprandStateMtgp32)));
        
    /* Allocate space for MTGP kernel parameters */
    CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
    
    /* Reformat from predefined parameter sets to kernel format, */
    /* and copy kernel parameters to device memory               */
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
    
    /* Initialize one state per thread block */
    CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, 
                mtgp32dc_params_fast_11213, devKernelParams, 64, 1234));
        
    /* Generate and use pseudo-random  */
    for(i = 0; i < 1; i++) {
        generate_kernel<<<1, 1>>>(devMTGPStates);
    }

    /* Cleanup */
    CUDA_CALL(hipFree(devMTGPStates));
    return EXIT_SUCCESS;
}